#include "hip/hip_runtime.h"
//
// Created by Shujian Qian on 2023-11-08.
//

#include <gpu_txn.cuh>
#include <gacco/benchmarks/tpcc_gpu_submitter.h>
#include <gacco/gpu_execution_planner.h>
#include <util_gpu_error_check.cuh>
#include <util_log.h>
#include <txn.h>

#include <hipcub/hipcub.hpp>

namespace gacco::tpcc {

using epic::BaseTxn;

TpccGpuSubmitter::TpccGpuSubmitter(TableSubmitDest warehouse_submit_dest, TableSubmitDest district_submit_dest,
    TableSubmitDest customer_submit_dest, TableSubmitDest history_submit_dest, TableSubmitDest new_order_submit_dest,
    TableSubmitDest order_submit_dest, TableSubmitDest order_line_submit_dest, TableSubmitDest item_submit_dest,
    TableSubmitDest stock_submit_dest, TpccConfig config)
    : TpccSubmitter(warehouse_submit_dest, district_submit_dest, customer_submit_dest, history_submit_dest,
          new_order_submit_dest, order_submit_dest, order_line_submit_dest, item_submit_dest, stock_submit_dest, config)
{
    for (int i = 0; i < 9; ++i)
    {
        hipStream_t stream;
        gpu_err_check(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
        cuda_streams.push_back(stream);
    }
}

TpccGpuSubmitter::~TpccGpuSubmitter()
{
    for (auto &stream : cuda_streams)
    {
        if (stream.has_value())
        {
            gpu_err_check(hipStreamDestroy(std::any_cast<hipStream_t>(stream)));
        }
    }
}

struct TpccNumOps
{
    uint32_t *warehouse_num_ops;
    uint32_t *district_num_ops;
    uint32_t *customer_num_ops;
    uint32_t *history_num_ops;
    uint32_t *order_num_ops;
    uint32_t *new_order_num_ops;
    uint32_t *order_line_num_ops;
    uint32_t *item_num_ops;
    uint32_t *stock_num_ops;
};

struct TpccSubmitLocations
{
    uint32_t *warehouse_offset;
    uint32_t *district_offset;
    uint32_t *customer_offset;
    uint32_t *history_offset;
    uint32_t *order_offset;
    uint32_t *new_order_offset;
    uint32_t *order_line_offset;
    uint32_t *item_offset;
    uint32_t *stock_offset;
    void *warehouse_dest;
    void *district_dest;
    void *customer_dest;
    void *history_dest;
    void *order_dest;
    void *new_order_dest;
    void *order_line_dest;
    void *item_dest;
    void *stock_dest;
};

static __device__ __forceinline__ void prepareSubmitTpccTxn(
    int txn_id, NewOrderTxnParams<FixedSizeTxn> *txn, TpccNumOps num_ops, TpccConfig config)
{
    if (config.gacco_use_atomic)
    {
        num_ops.warehouse_num_ops[txn_id] = 0;
        num_ops.district_num_ops[txn_id] = 0;
        num_ops.customer_num_ops[txn_id] = 0;
        num_ops.history_num_ops[txn_id] = 0;
        num_ops.order_num_ops[txn_id] = 0;
        num_ops.new_order_num_ops[txn_id] = 0;
        uint32_t num_items = txn->num_items;
        num_ops.order_line_num_ops[txn_id] = 0;
        num_ops.item_num_ops[txn_id] = 0;
        if (config.gacco_tpcc_stock_use_atomic)
        {
            num_ops.stock_num_ops[txn_id] = 0;
        }
        else
        {
            num_ops.stock_num_ops[txn_id] = num_items;
        }
    }
    else
    {
        num_ops.warehouse_num_ops[txn_id] = 1;
        num_ops.district_num_ops[txn_id] = 1;
        num_ops.customer_num_ops[txn_id] = 1;
        num_ops.history_num_ops[txn_id] = 0;
        num_ops.order_num_ops[txn_id] = 1;
        num_ops.new_order_num_ops[txn_id] = 1;
        uint32_t num_items = txn->num_items;
        num_ops.order_line_num_ops[txn_id] = num_items;
        num_ops.item_num_ops[txn_id] = num_items;
        num_ops.stock_num_ops[txn_id] = num_items;
    }
}

static __device__ __forceinline__ void prepareSubmitTpccTxn(
    int txn_id, PaymentTxnParams *txn, TpccNumOps num_ops, TpccConfig config)
{
    if (config.gacco_use_atomic)
    {
        num_ops.warehouse_num_ops[txn_id] = 0;
        num_ops.district_num_ops[txn_id] = 0;
        num_ops.customer_num_ops[txn_id] = 0;
    }
    else
    {
        num_ops.warehouse_num_ops[txn_id] = 1;
        num_ops.district_num_ops[txn_id] = 1;
        num_ops.customer_num_ops[txn_id] = 1;
    }
    num_ops.history_num_ops[txn_id] = 0; /* TODO: deal with history table later */
    num_ops.order_num_ops[txn_id] = 0;
    num_ops.new_order_num_ops[txn_id] = 0;
    num_ops.order_line_num_ops[txn_id] = 0;
    num_ops.item_num_ops[txn_id] = 0;
    num_ops.stock_num_ops[txn_id] = 0;
}

static __global__ void prepareSubmitTpccTxn(epic::GpuPackedTxnArray txn_array, TpccNumOps num_ops, TpccConfig config)
{
    int txn_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (txn_id >= txn_array.num_txns)
    {
        return;
    }
    BaseTxn *base_txn = txn_array.getTxn(txn_id);
    switch (static_cast<TpccTxnType>(base_txn->txn_type))
    {
    case TpccTxnType::NEW_ORDER:
        prepareSubmitTpccTxn(
            txn_id, reinterpret_cast<NewOrderTxnParams<FixedSizeTxn> *>(base_txn->data), num_ops, config);
        break;
    case TpccTxnType::PAYMENT:
        prepareSubmitTpccTxn(txn_id, reinterpret_cast<PaymentTxnParams *>(base_txn->data), num_ops, config);
        break;
    case TpccTxnType::ORDER_STATUS:
        /* TODO: implement prepare submit for order status */
        break;
    case TpccTxnType::DELIVERY:
        /* TODO: implement prepare submit for delivery */
        break;
    case TpccTxnType::STOCK_LEVEL:
        /* TODO: implement prepare submit for stock level */
        break;
    default:
        assert(false);
    }
}

static __device__ __forceinline__ void submitTpccTxn(
    int txn_id, NewOrderTxnParams<FixedSizeTxn> *txn, TpccSubmitLocations submit_loc, TpccConfig config)
{
    if (!config.gacco_use_atomic)
    {
        static_cast<uint64_t *>(submit_loc.warehouse_dest)[submit_loc.warehouse_offset[txn_id]] =
            GACCO_CREATE_OP(txn->warehouse_id, txn_id);
        static_cast<uint64_t *>(submit_loc.district_dest)[submit_loc.district_offset[txn_id]] =
            GACCO_CREATE_OP(txn->district_id, txn_id);
        static_cast<uint64_t *>(submit_loc.customer_dest)[submit_loc.customer_offset[txn_id]] =
            GACCO_CREATE_OP(txn->customer_id, txn_id);
        static_cast<uint64_t *>(submit_loc.order_dest)[submit_loc.order_offset[txn_id]] =
            GACCO_CREATE_OP(txn->order_id, txn_id);
        static_cast<uint64_t *>(submit_loc.new_order_dest)[submit_loc.new_order_offset[txn_id]] =
            GACCO_CREATE_OP(txn->new_order_id, txn_id);
        for (int i = 0; i < txn->num_items; i++)
        {
            static_cast<uint64_t *>(submit_loc.order_line_dest)[submit_loc.order_line_offset[txn_id] + i] =
                GACCO_CREATE_OP(txn->items[i].order_line_id, txn_id);
            static_cast<uint64_t *>(submit_loc.item_dest)[submit_loc.item_offset[txn_id] + i] =
                GACCO_CREATE_OP(txn->items[i].item_id, txn_id);
            static_cast<uint64_t *>(submit_loc.stock_dest)[submit_loc.stock_offset[txn_id] + i] =
                GACCO_CREATE_OP(txn->items[i].stock_id, txn_id);
        }
    }
    else if (!config.gacco_tpcc_stock_use_atomic)
    {
        for (int i = 0; i < txn->num_items; i++)
        {
            static_cast<uint64_t *>(submit_loc.stock_dest)[submit_loc.stock_offset[txn_id] + i] =
                GACCO_CREATE_OP(txn->items[i].stock_id, txn_id);
        }
    }
}

static __device__ __forceinline__ void submitTpccTxn(
    int txn_id, PaymentTxnParams *txn, TpccSubmitLocations submit_loc, TpccConfig config)
{
    if (!config.gacco_use_atomic)
    {
        static_cast<uint64_t *>(submit_loc.warehouse_dest)[submit_loc.warehouse_offset[txn_id]] =
            GACCO_CREATE_OP(txn->warehouse_id, txn_id);
        static_cast<uint64_t *>(submit_loc.district_dest)[submit_loc.district_offset[txn_id]] =
            GACCO_CREATE_OP(txn->district_id, txn_id);
        static_cast<uint64_t *>(submit_loc.customer_dest)[submit_loc.customer_offset[txn_id]] =
            GACCO_CREATE_OP(txn->customer_id, txn_id);
    }
}

static __global__ void submitTpccTxn(
    epic::GpuPackedTxnArray txn_array, TpccSubmitLocations submit_loc, TpccConfig config)
{
    int txn_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (txn_id >= txn_array.num_txns)
    {
        return;
    }
    BaseTxn *base_txn = txn_array.getTxn(txn_id);
    switch (static_cast<TpccTxnType>(base_txn->txn_type))
    {
    case TpccTxnType::NEW_ORDER:
        submitTpccTxn(txn_id, reinterpret_cast<NewOrderTxnParams<FixedSizeTxn> *>(base_txn->data), submit_loc, config);
        break;
    case TpccTxnType::PAYMENT:
        submitTpccTxn(txn_id, reinterpret_cast<PaymentTxnParams *>(base_txn->data), submit_loc, config);
        break;
    case TpccTxnType::ORDER_STATUS:
        /* TODO: implement submit for order status */
        break;
    case TpccTxnType::DELIVERY:
        /* TODO: implement submit for delivery */
        break;
    case TpccTxnType::STOCK_LEVEL:
        /* TODO: implement submit for stock level */
        break;
    default:
        assert(false);
    }
}

void TpccGpuSubmitter::submit(PackedTxnArray<TpccTxnParam> &txn_array)
{
    auto &logger = epic::Logger::GetInstance();

    TpccNumOps num_ops = {.warehouse_num_ops = warehouse_submit_dest.d_num_ops,
        .district_num_ops = district_submit_dest.d_num_ops,
        .customer_num_ops = customer_submit_dest.d_num_ops,
        .history_num_ops = history_submit_dest.d_num_ops,
        .order_num_ops = order_submit_dest.d_num_ops,
        .new_order_num_ops = new_order_submit_dest.d_num_ops,
        .order_line_num_ops = order_line_submit_dest.d_num_ops,
        .item_num_ops = item_submit_dest.d_num_ops,
        .stock_num_ops = stock_submit_dest.d_num_ops};

    prepareSubmitTpccTxn<<<(txn_array.num_txns + 1024) / 1024, 1024, 0, std::any_cast<hipStream_t>(cuda_streams[0])>>>(
        epic::GpuPackedTxnArray(txn_array), num_ops, config);

    gpu_err_check(hipGetLastError());
    gpu_err_check(hipStreamSynchronize(std::any_cast<hipStream_t>(cuda_streams[0])));

    gpu_err_check(hipcub::DeviceScan::InclusiveSum(warehouse_submit_dest.temp_storage,
        warehouse_submit_dest.temp_storage_bytes, warehouse_submit_dest.d_num_ops,
        warehouse_submit_dest.d_op_offsets + 1, txn_array.num_txns, std::any_cast<hipStream_t>(cuda_streams[0])));
    gpu_err_check(hipcub::DeviceScan::InclusiveSum(district_submit_dest.temp_storage,
        district_submit_dest.temp_storage_bytes, district_submit_dest.d_num_ops, district_submit_dest.d_op_offsets + 1,
        txn_array.num_txns, std::any_cast<hipStream_t>(cuda_streams[1])));
    gpu_err_check(hipcub::DeviceScan::InclusiveSum(customer_submit_dest.temp_storage,
        customer_submit_dest.temp_storage_bytes, customer_submit_dest.d_num_ops, customer_submit_dest.d_op_offsets + 1,
        txn_array.num_txns, std::any_cast<hipStream_t>(cuda_streams[2])));
    gpu_err_check(hipcub::DeviceScan::InclusiveSum(history_submit_dest.temp_storage,
        history_submit_dest.temp_storage_bytes, history_submit_dest.d_num_ops, history_submit_dest.d_op_offsets + 1,
        txn_array.num_txns, std::any_cast<hipStream_t>(cuda_streams[3])));
    gpu_err_check(hipcub::DeviceScan::InclusiveSum(order_submit_dest.temp_storage, order_submit_dest.temp_storage_bytes,
        order_submit_dest.d_num_ops, order_submit_dest.d_op_offsets + 1, txn_array.num_txns,
        std::any_cast<hipStream_t>(cuda_streams[4])));
    gpu_err_check(hipcub::DeviceScan::InclusiveSum(new_order_submit_dest.temp_storage,
        new_order_submit_dest.temp_storage_bytes, new_order_submit_dest.d_num_ops,
        new_order_submit_dest.d_op_offsets + 1, txn_array.num_txns, std::any_cast<hipStream_t>(cuda_streams[5])));
    gpu_err_check(hipcub::DeviceScan::InclusiveSum(order_line_submit_dest.temp_storage,
        order_line_submit_dest.temp_storage_bytes, order_line_submit_dest.d_num_ops,
        order_line_submit_dest.d_op_offsets + 1, txn_array.num_txns, std::any_cast<hipStream_t>(cuda_streams[6])));
    gpu_err_check(hipcub::DeviceScan::InclusiveSum(item_submit_dest.temp_storage, item_submit_dest.temp_storage_bytes,
        item_submit_dest.d_num_ops, item_submit_dest.d_op_offsets + 1, txn_array.num_txns,
        std::any_cast<hipStream_t>(cuda_streams[7])));
    gpu_err_check(hipcub::DeviceScan::InclusiveSum(stock_submit_dest.temp_storage, stock_submit_dest.temp_storage_bytes,
        stock_submit_dest.d_num_ops, stock_submit_dest.d_op_offsets + 1, txn_array.num_txns,
        std::any_cast<hipStream_t>(cuda_streams[8])));

    gpu_err_check(
        hipMemcpyAsync(&warehouse_submit_dest.curr_num_ops, warehouse_submit_dest.d_op_offsets + txn_array.num_txns,
            sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[0])));
    gpu_err_check(
        hipMemcpyAsync(&district_submit_dest.curr_num_ops, district_submit_dest.d_op_offsets + txn_array.num_txns,
            sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[1])));
    gpu_err_check(
        hipMemcpyAsync(&customer_submit_dest.curr_num_ops, customer_submit_dest.d_op_offsets + txn_array.num_txns,
            sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[2])));
    gpu_err_check(
        hipMemcpyAsync(&history_submit_dest.curr_num_ops, history_submit_dest.d_op_offsets + txn_array.num_txns,
            sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[3])));
    gpu_err_check(hipMemcpyAsync(&order_submit_dest.curr_num_ops, order_submit_dest.d_op_offsets + txn_array.num_txns,
        sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[4])));
    gpu_err_check(
        hipMemcpyAsync(&new_order_submit_dest.curr_num_ops, new_order_submit_dest.d_op_offsets + txn_array.num_txns,
            sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[5])));
    gpu_err_check(
        hipMemcpyAsync(&order_line_submit_dest.curr_num_ops, order_line_submit_dest.d_op_offsets + txn_array.num_txns,
            sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[6])));
    gpu_err_check(hipMemcpyAsync(&item_submit_dest.curr_num_ops, item_submit_dest.d_op_offsets + txn_array.num_txns,
        sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[7])));
    gpu_err_check(hipMemcpyAsync(&stock_submit_dest.curr_num_ops, stock_submit_dest.d_op_offsets + txn_array.num_txns,
        sizeof(uint32_t), hipMemcpyDeviceToHost, std::any_cast<hipStream_t>(cuda_streams[8])));

    TpccSubmitLocations locs = {
        .warehouse_offset = warehouse_submit_dest.d_op_offsets,
        .district_offset = district_submit_dest.d_op_offsets,
        .customer_offset = customer_submit_dest.d_op_offsets,
        .history_offset = history_submit_dest.d_op_offsets,
        .order_offset = order_submit_dest.d_op_offsets,
        .new_order_offset = new_order_submit_dest.d_op_offsets,
        .order_line_offset = order_line_submit_dest.d_op_offsets,
        .item_offset = item_submit_dest.d_op_offsets,
        .stock_offset = stock_submit_dest.d_op_offsets,
        .warehouse_dest = warehouse_submit_dest.d_submitted_ops,
        .district_dest = district_submit_dest.d_submitted_ops,
        .customer_dest = customer_submit_dest.d_submitted_ops,
        .history_dest = history_submit_dest.d_submitted_ops,
        .order_dest = order_submit_dest.d_submitted_ops,
        .new_order_dest = new_order_submit_dest.d_submitted_ops,
        .order_line_dest = order_line_submit_dest.d_submitted_ops,
        .item_dest = item_submit_dest.d_submitted_ops,
        .stock_dest = stock_submit_dest.d_submitted_ops,
    };

    submitTpccTxn<<<(txn_array.num_txns + 1024) / 1024, 1024, 0, std::any_cast<hipStream_t>(cuda_streams[0])>>>(
        epic::GpuPackedTxnArray(txn_array), locs, config);

    gpu_err_check(hipGetLastError());
    for (auto &stream : cuda_streams)
    {
        gpu_err_check(hipStreamSynchronize(std::any_cast<hipStream_t>(stream)));
    }

    logger.Info("num txns: {}", txn_array.num_txns);
    logger.Info("warehouse num ops: {}", warehouse_submit_dest.curr_num_ops);
    logger.Info("district num ops: {}", district_submit_dest.curr_num_ops);
    logger.Info("customer num ops: {}", customer_submit_dest.curr_num_ops);
    logger.Info("history num ops: {}", history_submit_dest.curr_num_ops);
    logger.Info("order num ops: {}", order_submit_dest.curr_num_ops);
    logger.Info("new order num ops: {}", new_order_submit_dest.curr_num_ops);
    logger.Info("order line num ops: {}", order_line_submit_dest.curr_num_ops);
    logger.Info("item num ops: {}", item_submit_dest.curr_num_ops);
    logger.Info("stock num ops: {}", stock_submit_dest.curr_num_ops);

#if 0 /* for debugging only */
    op_t ops[100];
    gpu_err_check(hipMemcpy(ops, warehouse_submit_dest.d_submitted_ops, sizeof(op_t) * 100, hipMemcpyDeviceToHost));
    for (int i = 0; i < 100; i++)
    {
        logger.Info("op{}: record[{}] txn[{}] rw[{}] offset[{}]", i, GET_RECORD_ID(ops[i]), GET_TXN_ID(ops[i]),
            GET_R_W(ops[i]), GET_OFFSET(ops[i]));
    }
#endif
}

} // namespace gacco::tpcc