//
// Created by Shujian Qian on 2023-10-08.
//

#include "util_gpu_transfer.h"

#include "util_gpu_error_check.cuh"

namespace epic {

std::any createGpuStream()
{
    hipStream_t stream;
    gpu_err_check(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    return stream;
}

void destroyGpuStream(std::any &stream)
{
    auto s = std::any_cast<hipStream_t>(stream);
    gpu_err_check(hipStreamDestroy(s));
    stream.reset();
}

void transferCpuToGpu(void *dst, const void *src, size_t size, std::any &stream)
{
    auto s = std::any_cast<hipStream_t>(stream);
    gpu_err_check(hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, s));
}

void transferCpuToGpu(void *dst, const void *src, size_t size)
{
    gpu_err_check(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void transferGpuToCpu(void *dst, const void *src, size_t size, std::any &stream)
{
    auto s = std::any_cast<hipStream_t>(stream);
    gpu_err_check(hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, s));
}

void transferGpuToCpu(void *dst, const void *src, size_t size)
{
    gpu_err_check(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

void syncGpuStream(std::any &stream)
{
    auto s = std::any_cast<hipStream_t>(stream);
    gpu_err_check(hipStreamSynchronize(s));
}

} // namespace epic