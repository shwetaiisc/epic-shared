//
// Created by Shujian Qian on 2023-08-09.
//

#include "gpu_allocator.h"

#include "util_gpu_error_check.cuh"
#include "util_log.h"
#include "util_math.h"

namespace epic {
void *GpuAllocator::Allocate(size_t size)
{
    void *ptr;
    gpu_err_check(hipMalloc(&ptr, size));
    gpu_err_check(hipMemset(ptr, 0, size));
    return ptr;
}

void GpuAllocator::Free(void *ptr)
{
    gpu_err_check(hipFree(ptr));
}

void GpuAllocator::PrintMemoryInfo()
{
    size_t free, total;
    gpu_err_check(hipMemGetInfo(&free, &total));
    auto &logger = Logger::GetInstance();
    logger.Info("GPU memory usage: {} / {}", formatSizeBytes(total - free), formatSizeBytes(total));
}
} // namespace epic