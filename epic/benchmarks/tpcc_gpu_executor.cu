#include "hip/hip_runtime.h"
//
// Created by Shujian Qian on 2023-10-25.
//

#include "tpcc_gpu_txn.cuh"

#include <benchmarks/tpcc_gpu_executor.h>

#include <stdio.h>

#include <gpu_storage.cuh>
#include <util_gpu_error_check.cuh>
#include <util_arch.h>
#include <gpu_txn.cuh>
#include <util_warp_memory.cuh>
#include <util_gpu_transfer.h>
#include <util_log.h>

namespace epic::tpcc {

namespace {

constexpr uint32_t block_size = 128;
static_assert(block_size % kDeviceWarpSize == 0, "block_size must be a multiple of 32");
constexpr uint32_t num_warps = block_size / kDeviceWarpSize;

__device__ uint32_t txn_counter = 0; /* used for scheduling txns among threads */
const uint32_t zero = 0;

__device__ __forceinline__ void gpuExecTpccTxn(TpccRecords records, TpccVersions versions,
    NewOrderTxnParams<FixedSizeTxn> *params, NewOrderExecPlan<FixedSizeTxn> *plan, uint32_t epoch, uint32_t lane_id,
    uint32_t txn_id /* for debug TODO: remove*/)
{
    constexpr uint32_t leader_lane = 0;
    constexpr uint32_t all_lanes_mask = 0xffffffffu;
    constexpr uint32_t s_quantity_offset = offsetof(StockValue, s_quantity) / sizeof(uint32_t);
    constexpr uint32_t d_next_o_id_offset = offsetof(DistrictValue, d_next_o_id) / sizeof(uint32_t);

#if 0 // DEBUG
    if (lane_id == leader_lane)
//        if (txn_id < 100 && lane_id == leader_lane)
    {
        printf("neworder txn_id[%u] warehouse i[%u]l[%u] district i[%u]l[%u] "
               "num_items[%u] "
               "item1[%u] stock_read1[%u] stock_write1[%u] orderline1_write[%u]"
               "\n",
            txn_id, params->warehouse_id, plan->warehouse_loc, params->district_id, plan->district_loc,
            params->num_items, plan->item_plans[0].item_loc, plan->item_plans[0].stock_read_loc,
            plan->item_plans[0].stock_write_loc, plan->item_plans[0].orderline_loc);
    }
#endif

    uint32_t result = 0;
    gpuReadFromTableCoop(records.warehouse_record, versions.warehouse_version, params->warehouse_id,
        plan->warehouse_loc, epoch, result, lane_id);

    gpuReadFromTableCoop(records.district_record, versions.district_version, params->district_id, plan->district_loc,
        epoch, result, lane_id);
    if (lane_id == d_next_o_id_offset)
    {
        // printf("RESULT[%u] tid[%u] district[%u] next_o_id[%u]\n", result, txn_id, params->district_id, params->next_order_id);
        result = params->next_order_id;
    }
    gpuWriteToTableCoop(records.district_record, versions.district_version, params->district_id,
        plan->district_write_loc, epoch, result, lane_id);

    gpuReadFromTableCoop(records.customer_record, versions.customer_version, params->customer_id, plan->customer_loc,
        epoch, result, lane_id);

    gpuWriteToTableCoop(
        records.order_record, versions.order_version, params->order_id, plan->order_loc, epoch, result, lane_id);

    gpuWriteToTableCoop(records.new_order_record, versions.new_order_version, params->new_order_id, plan->new_order_loc,
        epoch, result, lane_id);

    for (uint32_t i = 0; i < params->num_items; ++i)
    {
        gpuReadFromTableCoop(records.item_record, versions.item_version, params->items[i].item_id,
            plan->item_plans[i].item_loc, epoch, result, lane_id);
        gpuReadFromTableCoop(records.stock_record, versions.stock_version, params->items[i].stock_id,
            plan->item_plans[i].stock_read_loc, epoch, result, lane_id);
        if (lane_id == s_quantity_offset)
        {
            uint32_t order_quantity = params->items[i].order_quantities;
            result = result > order_quantity + 10 ? result - order_quantity : result + 91 - order_quantity;
        }
        gpuWriteToTableCoop(records.stock_record, versions.stock_version, params->items[i].stock_id,
            plan->item_plans[i].stock_write_loc, epoch, result, lane_id);

        constexpr uint32_t ol_i_id_offset = offsetof(OrderLineValue, ol_i_id) / sizeof(uint32_t);
        constexpr uint32_t ol_amount_offset = offsetof(OrderLineValue, ol_amount) / sizeof(uint32_t);
        constexpr uint32_t ol_supply_w_id_offset = offsetof(OrderLineValue, ol_supply_w_id) / sizeof(uint32_t);
        constexpr uint32_t ol_quantity_offset = offsetof(OrderLineValue, ol_quantity) / sizeof(uint32_t);
        if (lane_id == ol_i_id_offset)
        {
            result = params->items[i].item_id;
        }
        if (lane_id == ol_amount_offset)
        {
            result = params->items[i].order_quantities;
        }
        if (lane_id == ol_supply_w_id_offset)
        {
            result = params->warehouse_id;
        }
        if (lane_id == ol_quantity_offset)
        {
            result = params->items[i].order_quantities;
        }
        gpuWriteToTableCoop(records.order_line_record, versions.order_line_version, params->items[i].order_line_id,
            plan->item_plans[i].orderline_loc, epoch, result, lane_id);
    }
}

__device__ __forceinline__ void gpuExecTpccTxn(TpccRecords records, TpccVersions versions, PaymentTxnParams *params,
    PaymentTxnExecPlan *plan, uint32_t epoch, uint32_t lane_id, uint32_t txn_id /* for debug TODO: remove*/)
{
    constexpr uint32_t leader_lane = 0;
    constexpr uint32_t all_lanes_mask = 0xffffffffu;
    constexpr uint32_t w_ytd_offset = offsetof(WarehouseValue, w_ytd) / sizeof(uint32_t);
    constexpr uint32_t d_ytd_offset = offsetof(DistrictValue, d_ytd) / sizeof(uint32_t);
    constexpr uint32_t c_balance_offset = offsetof(CustomerValue, c_balance) / sizeof(uint32_t);
    constexpr uint32_t c_ytd_payment_offset = offsetof(CustomerValue, c_ytd_payment) / sizeof(uint32_t);
    constexpr uint32_t c_payment_cnt_offset = offsetof(CustomerValue, c_payment_cnt) / sizeof(uint32_t);

#if 0 // DEBUG
    {
//                if (lane_id == leader_lane && txn_id < 100)
        if (lane_id == leader_lane)
        {
            printf("payment txn_id[%u] warehouse i[%u]rl[%u]wl[%u] district i[%u]rl[%u]wl[%u] customer i[%u]rl[%u]wl[%u] "
                   "payment_amount[%u]\n",
                txn_id, params->warehouse_id, plan->warehouse_read_loc, plan->warehouse_write_loc, params->district_id,
                plan->district_read_loc, plan->district_write_loc, params->customer_id, plan->customer_read_loc,
                plan->customer_write_loc, params->payment_amount);
        }
    }
#endif

    uint32_t result;
    uint32_t payment_amount = params->payment_amount;

    gpuReadFromTableCoop(records.warehouse_record, versions.warehouse_version, params->warehouse_id,
        plan->warehouse_read_loc, epoch, result, lane_id);
    if (lane_id == w_ytd_offset)
    {
        result += payment_amount;
    }
    gpuWriteToTableCoop(records.warehouse_record, versions.warehouse_version, params->warehouse_id,
        plan->warehouse_write_loc, epoch, result, lane_id);

    gpuReadFromTableCoop(records.district_record, versions.district_version, params->district_id,
        plan->district_read_loc, epoch, result, lane_id);
    if (lane_id == d_ytd_offset)
    {
        result += payment_amount;
    }
    gpuWriteToTableCoop(records.district_record, versions.district_version, params->district_id,
        plan->district_write_loc, epoch, result, lane_id);

    gpuReadFromTableCoop(records.customer_record, versions.customer_version, params->customer_id,
        plan->customer_read_loc, epoch, result, lane_id);
    if (lane_id == c_balance_offset)
    {
        result -= payment_amount;
    }
    if (lane_id == c_ytd_payment_offset)
    {
        result += payment_amount;
    }
    if (lane_id == c_payment_cnt_offset)
    {
        result += 1;
    }
    gpuWriteToTableCoop(records.customer_record, versions.customer_version, params->customer_id,
        plan->customer_write_loc, epoch, result, lane_id);
}

__device__ __forceinline__ void gpuExecTpccTxn(TpccRecords records, TpccVersions versions, OrderStatusTxnParams *params,
    OrderStatusTxnExecPlan *plan, uint32_t epoch, uint32_t lane_id, uint32_t txn_id /* for debug TODO: remove*/)
{
    uint32_t result;
    gpuReadFromTableCoop(records.customer_record, versions.customer_version, params->customer_id, plan->customer_loc,
        epoch, result, lane_id);
    gpuReadFromTableCoop(
        records.order_record, versions.order_version, params->order_id, plan->order_loc, epoch, result, lane_id);
    for (int i = 0; i < params->num_items; ++i)
    {
        gpuReadFromTableCoop(records.order_line_record, versions.order_line_version, params->orderline_ids[i],
            plan->orderline_locs[i], epoch, result, lane_id);
    }
}

void __device__ __forceinline__ gpuExecTpccTxn(TpccRecords records, TpccVersions versions, DeliveryTxnParams *params,
    DeliveryTxnExecPlan *plan, uint32_t epoch, uint32_t lane_id, uint32_t txn_id)
{
    uint32_t result;
    for (int i = 0; i < 10; ++i)
    {
        gpuReadFromTableCoop(records.new_order_record, versions.new_order_version, params->new_order_id[i],
            plan->new_order_read_locs[i], epoch, result, lane_id);

        constexpr uint32_t o_carrier_id_offset = offsetof(OrderValue, o_carrier_id) / sizeof(uint32_t);
        gpuReadFromTableCoop(records.order_record, versions.order_version, params->order_id[i],
            plan->order_read_locs[i], epoch, result, lane_id);

        if (lane_id == o_carrier_id_offset)
        {
            result = params->carrier_id;
        }
        gpuWriteToTableCoop(records.order_record, versions.order_version, params->order_id[i],
            plan->order_write_locs[i], epoch, result, lane_id);

        constexpr uint32_t ol_amount_offset = offsetof(OrderLineValue, ol_amount) / sizeof(uint32_t);
        constexpr uint32_t ol_delivery_d_offset = offsetof(OrderLineValue, ol_delivery_d) / sizeof(uint32_t);
        uint32_t amount = 0;
        for (int j = 0; j < params->num_items[i]; ++j)
        {
            gpuReadFromTableCoop(records.order_line_record, versions.order_line_version, params->orderline_ids[i][j],
                plan->orderline_read_locs[i][j], epoch, result, lane_id);
            if (lane_id == ol_amount_offset)
            {
                amount += result;
            }
            if (lane_id == ol_delivery_d_offset)
            {
                result = params->delivery_d;
            }

            gpuWriteToTableCoop(records.order_line_record, versions.order_line_version, params->orderline_ids[i][j],
                loc_record_b, epoch, result, lane_id);
        }

        constexpr uint32_t all_lanes_mask = 0xffffffffu;
        __shfl_sync(all_lanes_mask, amount, ol_amount_offset);

        gpuReadFromTableCoop(records.customer_record, versions.customer_version, params->customer_id[i],
            plan->customer_read_locs[i], epoch, result, lane_id);

        constexpr uint32_t c_balance_offset = offsetof(CustomerValue, c_balance) / sizeof(uint32_t);
        constexpr uint32_t c_delivery_cnt_offset = offsetof(CustomerValue, c_delivery_cnt) / sizeof(uint32_t);

        if (lane_id == c_balance_offset)
        {
            result += amount;
        }
        if (lane_id == c_delivery_cnt_offset)
        {
            ++result;
        }

        gpuWriteToTableCoop(records.customer_record, versions.customer_version, params->customer_id[i],
            plan->customer_write_locs[i], epoch, result, lane_id);

    }
}

void __device__ __forceinline__ gpuExecTpccTxn(TpccRecords records, TpccVersions versions, StockLevelTxnParams *params,
    StockLevelTxnExecPlan *plan, uint32_t epoch, uint32_t lane_id, uint32_t txn_id)
{
    uint32_t num_low_stock = 0;
    const uint32_t threshold = params->threshold;
    uint32_t result;
    constexpr uint32_t s_quantity_offset = offsetof(StockValue, s_quantity) / sizeof(uint32_t);
    for (uint32_t i = 0; i < params->num_items; ++i)
    {
        gpuReadFromTableCoop(records.stock_record, versions.stock_version, params->stock_ids[i],
            plan->stock_read_locs[i], epoch, result, lane_id);
        if (lane_id == s_quantity_offset && result < threshold)
        {
            ++num_low_stock;
        }
    }
    if (lane_id == s_quantity_offset)
    {
        params->num_low_stock = num_low_stock;
    }
}

union CachableTxnParams
{
    NewOrderTxnParams<FixedSizeTxn> no;
    PaymentTxnParams pmt;
    OrderStatusTxnParams os;
} __attribute__((aligned(4)));

union CachableTxnExecPlan
{
    NewOrderExecPlan<FixedSizeTxn> no;
    PaymentTxnExecPlan pmt;
    OrderStatusTxnExecPlan os;
} __attribute__((aligned(4)));

static_assert(sizeof(CachableTxnExecPlan) + sizeof(CachableTxnParams) < 1000);

template <typename GpuTxnArrayType>
__global__ void gpuExecKernel(TpccRecords records, TpccVersions versions, GpuTxnArrayType txn, GpuTxnArrayType plan,
    uint32_t num_txns, uint32_t epoch)
{
    constexpr uint32_t leader_lane = 0;
    constexpr uint32_t all_lanes_mask = 0xffffffffu;

    __shared__ uint8_t cached_txn_param[num_warps][BaseTxnSize<CachableTxnParams>::value];
    __shared__ uint8_t cached_exec_plan[num_warps][BaseTxnSize<CachableTxnExecPlan>::value];
    static_assert(BaseTxnSize<CachableTxnParams>::value % sizeof(uint32_t) == 0, "Cannot be copied in 32-bit words");
    static_assert(BaseTxnSize<CachableTxnExecPlan>::value % sizeof(uint32_t) == 0, "Cannot be copied in 32-bit words");
    __shared__ uint32_t warp_counter;

    uint32_t warp_id = threadIdx.x / kDeviceWarpSize;
    uint32_t lane_id = threadIdx.x % kDeviceWarpSize;
    /* one thread loads txn id for the entire warp */
    if (threadIdx.x == 0)
    {
        warp_counter = atomicAdd(&txn_counter, num_warps);
    }

    __syncthreads();
    /* warp cooperative execution afterward */

    uint32_t warp_txn_id;
    if (lane_id == leader_lane)
    {
        warp_txn_id = atomicAdd(&warp_counter, 1);
    }
    warp_txn_id = __shfl_sync(all_lanes_mask, warp_txn_id, leader_lane);
    if (warp_txn_id >= num_txns)
    {
        return;
    }

    /* load txn param and exec plan into shared memory */
    BaseTxn *txn_param_ptr = txn.getTxn(warp_txn_id);
    BaseTxn *exec_plan_ptr = plan.getTxn(warp_txn_id);

    /* execute the txn */
    switch (static_cast<TpccTxnType>((reinterpret_cast<BaseTxn *>(txn_param_ptr)->txn_type)))
    {
    case TpccTxnType::NEW_ORDER:
        warpMemcpy(reinterpret_cast<uint32_t *>(cached_txn_param[warp_id]), reinterpret_cast<uint32_t *>(txn_param_ptr),
            BaseTxnSize<NewOrderTxnParams<FixedSizeTxn>>::value / sizeof(uint32_t), lane_id);
        warpMemcpy(reinterpret_cast<uint32_t *>(cached_exec_plan[warp_id]), reinterpret_cast<uint32_t *>(exec_plan_ptr),
            BaseTxnSize<NewOrderExecPlan<FixedSizeTxn>>::value / sizeof(uint32_t), lane_id);
        __syncwarp();
        gpuExecTpccTxn(records, versions,
            reinterpret_cast<NewOrderTxnParams<FixedSizeTxn> *>(
                reinterpret_cast<BaseTxn *>(cached_txn_param[warp_id])->data),
            reinterpret_cast<NewOrderExecPlan<FixedSizeTxn> *>(
                reinterpret_cast<BaseTxn *>(cached_exec_plan[warp_id])->data),
            epoch, lane_id, warp_txn_id);
        break;
    case TpccTxnType::PAYMENT:
        warpMemcpy(reinterpret_cast<uint32_t *>(cached_txn_param[warp_id]), reinterpret_cast<uint32_t *>(txn_param_ptr),
            BaseTxnSize<PaymentTxnParams>::value / sizeof(uint32_t), lane_id);
        warpMemcpy(reinterpret_cast<uint32_t *>(cached_exec_plan[warp_id]), reinterpret_cast<uint32_t *>(exec_plan_ptr),
            BaseTxnSize<PaymentTxnExecPlan>::value / sizeof(uint32_t), lane_id);
        __syncwarp();
        gpuExecTpccTxn(records, versions,
            reinterpret_cast<PaymentTxnParams *>(reinterpret_cast<BaseTxn *>(cached_txn_param[warp_id])->data),
            reinterpret_cast<PaymentTxnExecPlan *>(reinterpret_cast<BaseTxn *>(cached_exec_plan[warp_id])->data), epoch,
            lane_id, warp_txn_id);
        break;
    case TpccTxnType::ORDER_STATUS:
        warpMemcpy(reinterpret_cast<uint32_t *>(cached_txn_param[warp_id]), reinterpret_cast<uint32_t *>(txn_param_ptr),
            BaseTxnSize<OrderStatusTxnParams>::value / sizeof(uint32_t), lane_id);
        warpMemcpy(reinterpret_cast<uint32_t *>(cached_exec_plan[warp_id]), reinterpret_cast<uint32_t *>(exec_plan_ptr),
            BaseTxnSize<OrderStatusTxnExecPlan>::value / sizeof(uint32_t), lane_id);
        __syncwarp();
        gpuExecTpccTxn(records, versions,
            reinterpret_cast<OrderStatusTxnParams *>(reinterpret_cast<BaseTxn *>(cached_txn_param[warp_id])->data),
            reinterpret_cast<OrderStatusTxnExecPlan *>(reinterpret_cast<BaseTxn *>(cached_exec_plan[warp_id])->data),
            epoch, lane_id, warp_txn_id);
        break;
    case TpccTxnType::DELIVERY:
        gpuExecTpccTxn(records, versions, reinterpret_cast<DeliveryTxnParams *>(txn_param_ptr->data),
            reinterpret_cast<DeliveryTxnExecPlan *>(exec_plan_ptr->data), epoch, lane_id, warp_txn_id);
            break;
    case TpccTxnType::STOCK_LEVEL:
        gpuExecTpccTxn(records, versions, reinterpret_cast<StockLevelTxnParams *>(txn_param_ptr->data),
            reinterpret_cast<StockLevelTxnExecPlan *>(exec_plan_ptr->data), epoch, lane_id, warp_txn_id);
        break;
    default:
        assert(false);
        break;
    }
}

} // namespace

template <typename TxnParamArrayType, typename TxnExecPlanArrayType>
void GpuExecutor<TxnParamArrayType, TxnExecPlanArrayType>::execute(uint32_t epoch)
{
    /* clear the txn_counter */
    gpu_err_check(hipMemcpyToSymbol(HIP_SYMBOL(txn_counter), &zero, sizeof(uint32_t)));

#if 0 // DEBUG
    {
        auto &logger = Logger::GetInstance();
        constexpr size_t max_print_size = 100u;
        constexpr size_t base_txn_size = TxnArray<TpccTxnParam>::kBaseTxnSize;
        uint32_t print_size = std::min(config.num_txns, max_print_size);
        uint32_t copy_size = print_size * base_txn_size;
        uint8_t txn_params[max_print_size * base_txn_size];

        transferGpuToCpu(txn_params, txn.txns, copy_size);
        for (int i = 0; i < print_size; ++i)
        {
            auto param = &reinterpret_cast<TpccTxnParam *>(
                reinterpret_cast<BaseTxn *>(txn_params + i * base_txn_size)->data)
                ->new_order_txn;
            logger.Info("txn {} warehouse[{}] district[{}] customer[{}] order[{}] new_order[{}] numitems[{}] "
                        "item1[{}] stock_read1[{}] order_line1[{}] quantity1[{}] "
                        "item2[{}] stock_read2[{}] order_line2[{}] quantity2[{}] "
                        "item3[{}] stock_read3[{}] order_line3[{}] quantity3[{}] "
                        "item4[{}] stock_read4[{}] order_line4[{}] quantity4[{}] "
                        "item5[{}] stock_read5[{}] order_line5[{}] quantity5[{}] ",
                        i, param->warehouse_id, param->district_id, param->customer_id, param->order_id,
                        param->new_order_id, param->num_items, param->items[0].item_id, param->items[0].stock_id,
                        param->items[0].order_line_id, param->items[0].order_quantities, param->items[1].item_id,
                        param->items[1].stock_id, param->items[1].order_line_id, param->items[1].order_quantities,
                        param->items[2].item_id, param->items[2].stock_id, param->items[2].order_line_id,
                        param->items[2].order_quantities, param->items[3].item_id, param->items[3].stock_id,
                        param->items[3].order_line_id, param->items[3].order_quantities, param->items[4].item_id,
                        param->items[4].stock_id, param->items[4].order_line_id, param->items[4].order_quantities);
        }
        logger.flush();
    }
#endif

    uint32_t num_blocks = (config.num_txns * kDeviceWarpSize + block_size - 1) / block_size;
    gpuExecKernel<<<num_blocks, block_size>>>(
        records, versions, TpccGpuTxnArrayT(txn), TpccGpuTxnArrayT(plan), config.num_txns, epoch);
    gpu_err_check(hipPeekAtLastError());
    gpu_err_check(hipDeviceSynchronize());
}

template class GpuExecutor<TpccTxnParamArrayT, TpccTxnExecPlanArrayT>;

} // namespace epic::tpcc