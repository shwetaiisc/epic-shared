#include "hip/hip_runtime.h"

/*
 *   Copyright 2022 The Regents of the University of California, Davis
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 */

#include <gpu_btree.h>
#include <gtest/gtest.h>
#include <cmd.hpp>
#include <cstdint>
#include <random>
#include <unordered_set>
#include <map>
#include <chrono>

std::size_t num_keys = 30000;

namespace {
using key_type   = uint32_t;
using value_type = uint32_t;

union PackedCustomerOrderKey {
  using baseType = uint64_t;
  constexpr static baseType max_o_id = (1ull << 20) - 1;
  constexpr static baseType invalid_key = (1ull << 42) - 1;
  struct {
    baseType o_id : 20;
    baseType c_id : 12;
    baseType d_id : 4;
    baseType w_id : 6;
  };
  baseType base_key        = 0;
  PackedCustomerOrderKey() = default;
  PackedCustomerOrderKey(baseType o_id, baseType c_id, baseType d_id, baseType w_id) {
    base_key        = 0;
    this->o_id   = o_id;
    this->c_id   = c_id;
    this->d_id   = d_id;
    this->w_id = w_id;
  }
};

class RandomKeyGen {
  std::mt19937 gen;
  std::uniform_int_distribution<uint64_t> w_id_dist, d_id_dist, c_id_dist, o_id_dist, val_dist;

 public:
  RandomKeyGen()
      : gen(std::random_device{}())
      , w_id_dist(1, 64)
      , d_id_dist(1, 10)
      , c_id_dist(1, 3000)
      , o_id_dist(0, PackedCustomerOrderKey::max_o_id - 1)
      , val_dist(0, (1ull << 22) - 2) {}
  PackedCustomerOrderKey next() {
    return PackedCustomerOrderKey{o_id_dist(gen), c_id_dist(gen), d_id_dist(gen), w_id_dist(gen)};
  }
  uint64_t rand_val() { return val_dist(gen); }
};

TEST(PackedCustomerOrderKeyTest, TestLayout) {
  PackedCustomerOrderKey key;
  key.o_id = (1ull << 20) - 1;
  EXPECT_EQ(key.base_key, (1ull << 20) - 1);
  key.c_id = (1ull << 12) - 1;
  EXPECT_EQ(key.base_key, (1ull << 32) - 1);
  key.d_id = (1ull << 4) - 1;
  EXPECT_EQ(key.base_key, (1ull << 36) - 1);
  key.w_id = (1ull << 6) - 1;
  EXPECT_EQ(key.base_key, (1ull << 42) - 1);
}

// const auto sentinel_value = std::numeric_limits<key_type>::max();
// const auto sentinel_key = std::numeric_limits<value_type>::max();
template <typename BTreeMap>
struct BTreeMapData {
  using btree_map = BTreeMap;
};

template <typename T>
struct mapped_vector {
  mapped_vector(std::size_t capacity) : capacity_(capacity) { allocate(capacity); }
  T& operator[](std::size_t index) { return dh_buffer_[index]; }
  ~mapped_vector() {}
  void free() {
    cuda_try(hipDeviceSynchronize());
    cuda_try(hipHostFree(dh_buffer_));
  }
  T* data() const { return dh_buffer_; }

  std::vector<T> to_std_vector() {
    std::vector<T> copy(capacity_);
    for (std::size_t i = 0; i < capacity_; i++) { copy[i] = dh_buffer_[i]; }
    return copy;
  }

 private:
  void allocate(std::size_t count) { cuda_try(hipHostMalloc(&dh_buffer_, sizeof(T) * count)); }
  std::size_t capacity_;
  T* dh_buffer_;
};

template <typename key_type = uint32_t, typename value_type = uint32_t>
struct testing_input {
  testing_input(std::size_t input_num_keys)
      : num_keys(input_num_keys)
      , keys(input_num_keys)
      , values(input_num_keys)
      , keys_exist(input_num_keys)
      , keys_not_exist(input_num_keys)
      , max_keys(input_num_keys)
    , non_exist_keys_upper_bound(input_num_keys)
  {
    make_input();
  }
  void make_input() {
    RandomKeyGen key_gen;
    for (std::size_t i = 0; i < num_keys; i++) {
      // Make sure that the input doesn't contain 0
      // and, queries that do not exist in the table are uniformly distributed to avoid
      // contention... an optimzation is to avoid locking if key doesn't exist in node
      PackedCustomerOrderKey key;
      do {
        key = key_gen.next();
      } while (existing_keys.count(key.base_key) > 0);
      value_type val = key_gen.rand_val();
      existing_keys[key.base_key] = val;
      keys[i] = key.base_key;
      keys_exist[i] = key.base_key;
      values[i] = val;

      // upper bound with the same customer
      key.o_id = PackedCustomerOrderKey::max_o_id;
      max_keys[i] = key.base_key;
    }

    for (std::size_t i = 0; i < num_keys; i++) {
      // Make sure that the input doesn't contain 0
      // and, queries that do not exist in the table are uniformly distributed to avoid
      // contention... an optimzation is to avoid locking if key doesn't exist in node
      PackedCustomerOrderKey key;
      do {
        key = key_gen.next();
      } while (existing_keys.count(key.base_key) > 0 || non_existing_keys.count(key.base_key) > 0);
      keys_not_exist[i] = key.base_key;

      // upper bound with the same customer
      key.o_id = PackedCustomerOrderKey::max_o_id;
      non_exist_keys_upper_bound[i] = key.base_key;
    }
  }
  void free() {
    keys.free();
    values.free();
    keys_exist.free();
    keys_not_exist.free();
    max_keys.free();
  }

  std::size_t num_keys;
  mapped_vector<key_type> keys;
  mapped_vector<value_type> values;
  mapped_vector<key_type> keys_exist;
  mapped_vector<key_type> keys_not_exist;
  mapped_vector<key_type> max_keys;;
  mapped_vector<key_type> non_exist_keys_upper_bound;;
  std::map<key_type, value_type> existing_keys;
  std::unordered_set<key_type> non_existing_keys;
};

template <class MapData>
class BTreeMapTest : public testing::Test {
 protected:
  BTreeMapTest() { btree_map_ = new typename map_data::btree_map(); }
  ~BTreeMapTest() override { delete btree_map_; }
  using map_data         = MapData;
  using input_key_type   = typename map_data::btree_map::key_type;
  using input_value_type = typename map_data::btree_map::value_type;
  using input_pair_type = typename map_data::btree_map::pair_type;
  typename map_data::btree_map* btree_map_;
  testing_input<input_key_type, input_value_type> input{num_keys};
  mapped_vector<input_value_type> find_results{num_keys};
  mapped_vector<input_pair_type> find_pair_results{num_keys};
  constexpr static input_value_type sentinel_value = map_data::btree_map::pair_type::invalid_value;
};

struct TreeParam {
  static constexpr int BranchingFactor = 16;
};
struct SlabAllocParam {
  static constexpr uint32_t NumSuperBlocks  = 1;
  static constexpr uint32_t NumMemoryBlocks = 1024 * 1;
  static constexpr uint32_t TileSize        = TreeParam::BranchingFactor;
  static constexpr uint32_t SlabSize        = 128;
};

using node_type = GpuBTree::node_type<key_type,
                                      value_type,
                                      TreeParam::BranchingFactor,
                                      var_pair_type<uint64_t, uint64_t, 42, 22>>;
static_assert(sizeof(node_type) == 128);

using bump_allocator_type = device_bump_allocator<node_type, 100000>;
using slab_allocator_type = device_allocator::SlabAllocLight<node_type,
                                                             SlabAllocParam::NumSuperBlocks,
                                                             SlabAllocParam::NumMemoryBlocks,
                                                             SlabAllocParam::TileSize,
                                                             SlabAllocParam::SlabSize>;

typedef testing::Types<
    BTreeMapData<GpuBTree::gpu_blink_tree<uint64_t,
                                          uint64_t,
                                          TreeParam::BranchingFactor,
                                          bump_allocator_type,
                                          var_pair_type<uint64_t, uint64_t, 42, 22>>>,
    BTreeMapData<GpuBTree::gpu_blink_tree<uint64_t,
                                          uint64_t,
                                          TreeParam::BranchingFactor,
                                          slab_allocator_type,
                                          var_pair_type<uint64_t, uint64_t, 42, 22>>>
                                          >
    Implementations;

TYPED_TEST_SUITE(BTreeMapTest, Implementations);

TYPED_TEST(BTreeMapTest, Validation) {
  //  testing_input<input_key_type, input_value_type> input(num_keys);
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  auto keys = this->input.keys.to_std_vector();
  EXPECT_NO_THROW(this->btree_map_->validate_tree_structure(
      keys, [&](auto key) { return this->input.existing_keys[key]; }));
}

TYPED_TEST(BTreeMapTest, FindExist) {
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->find(this->input.keys_exist.data(), this->find_results.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  for (std::size_t i = 0; i < num_keys; i++) {
    auto expected_value = this->input.values[i];
    auto found_value    = this->find_results[i];
    ASSERT_EQ(found_value, expected_value);
  }
}

TYPED_TEST(BTreeMapTest, FindNotExist) {
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->find(this->input.keys_not_exist.data(), this->find_results.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  for (std::size_t i = 0; i < num_keys; i++) {
    auto expected_value = this->sentinel_value;
    auto found_value    = this->find_results[i];
    ASSERT_EQ(found_value, expected_value);
  }
}

TYPED_TEST(BTreeMapTest, EraseAllTest) {
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->erase(this->input.keys.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->find(this->input.keys_exist.data(), this->find_results.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  for (std::size_t i = 0; i < num_keys; i++) {
    auto expected_value = this->sentinel_value;
    auto found_value    = this->find_results[i];
    ASSERT_EQ(found_value, expected_value);
  }
}

TYPED_TEST(BTreeMapTest, EraseNoneTest) {
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->erase(this->input.keys_not_exist.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->find(this->input.keys_exist.data(), this->find_results.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  // EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  for (std::size_t i = 0; i < num_keys; i++) {
    auto expected_value = this->input.values[i];
    auto found_value    = this->find_results[i];
    ASSERT_EQ(found_value, expected_value);
  }
}

TYPED_TEST(BTreeMapTest, EraseAllInsertAllTest) {
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->erase(this->input.keys.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->find(this->input.keys_exist.data(), this->find_results.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  for (std::size_t i = 0; i < num_keys; i++) {
    auto expected_value = this->sentinel_value;
    auto found_value    = this->find_results[i];
    ASSERT_EQ(found_value, expected_value);
  }
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  this->btree_map_->find(this->input.keys_exist.data(), this->find_results.data(), num_keys);
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  cuda_try(hipDeviceSynchronize());
  for (std::size_t i = 0; i < num_keys; i++) {
    auto expected_value = this->input.values[i];
    auto found_value    = this->find_results[i];
    ASSERT_EQ(found_value, expected_value);
  }
}

TYPED_TEST(BTreeMapTest, FindNextExistTest) {
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  auto start = std::chrono::high_resolution_clock::now();
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  auto end = std::chrono::high_resolution_clock::now();
  std::cout << "Inserting " << num_keys << " took "
            << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() << "us" << std::endl;
  this->btree_map_->find_next(this->input.keys_exist.data(),
                              this->input.max_keys.data(),
                              this->find_pair_results.data(),
                              num_keys);
  cuda_try(hipDeviceSynchronize());
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  for (std::size_t i = 0; i < num_keys; i++) {
    auto expected_key = this->input.keys[i];
    auto found_key    = this->find_pair_results[i].first;
    EXPECT_EQ(found_key, expected_key);

    auto expected_value = this->input.values[i];
    auto found_value    = this->find_pair_results[i].second;
    EXPECT_EQ(found_value, expected_value);
  }
}

TYPED_TEST(BTreeMapTest, FindNextNonExistTest) {
  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);
  auto start = std::chrono::high_resolution_clock::now();
  this->btree_map_->insert(this->input.keys.data(), this->input.values.data(), num_keys);
  cuda_try(hipDeviceSynchronize());
  auto end = std::chrono::high_resolution_clock::now();
  std::cout << "Inserting " << num_keys << " took "
            << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() << "us" << std::endl;

  start = std::chrono::high_resolution_clock::now();
  this->btree_map_->find_next(this->input.keys_not_exist.data(),
                              this->input.non_exist_keys_upper_bound.data(),
                              this->find_pair_results.data(),
                              num_keys);
  cuda_try(hipDeviceSynchronize());
  end = std::chrono::high_resolution_clock::now();
  std::cout << "Finding Next for " << num_keys << " took "
            << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() << "us" << std::endl;

  EXPECT_EQ(hipDeviceSynchronize(), hipSuccess);

  uint64_t num_found = 0;
  for (std::size_t i = 0; i < num_keys; i++) {
    PackedCustomerOrderKey expect_key;
    expect_key.base_key = PackedCustomerOrderKey::invalid_key;
    constexpr value_type invalid_value = (1ull << 22) - 1;
    value_type expected_value = invalid_value;

    PackedCustomerOrderKey lowerbound_key;
    lowerbound_key.base_key = this->input.keys_not_exist[i];

    auto it = this->input.existing_keys.lower_bound(lowerbound_key.base_key);
    if (it != this->input.existing_keys.end()) {
      expect_key.base_key = it->first;

      // looking for orders of the same customer
      // if not the same customer, should return invalid kv pair
      if (expect_key.w_id != lowerbound_key.w_id
        || expect_key.d_id != lowerbound_key.d_id
        || expect_key.c_id != lowerbound_key.c_id) {
        expect_key.base_key = PackedCustomerOrderKey::invalid_key;
      } else {
        expected_value = this->input.existing_keys[expect_key.base_key];
        ++num_found;
      }
    }

    auto expected_key = expect_key.base_key;
    auto found_key    = this->find_pair_results[i].first;
    EXPECT_EQ(found_key, expected_key);

    auto found_value    = this->find_pair_results[i].second;
    EXPECT_EQ(found_value, expected_value);
  }
  std::cout << "Number of keys found for range_find_next: " << num_found << std::endl;
}

}  // namespace

int main(int argc, char** argv) {
  auto arguments = std::vector<std::string>(argv, argv + argc);
  num_keys       = get_arg_value<uint32_t>(arguments, "num-keys").value_or(500000);
  std::cout << "Testing using " << num_keys << " keys\n";
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}