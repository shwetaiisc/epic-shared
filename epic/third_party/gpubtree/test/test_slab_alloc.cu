#include "hip/hip_runtime.h"
/*
 *   Copyright 2022 The Regents of the University of California, Davis
 *
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *   you may not use this file except in compliance with the License.
 *   You may obtain a copy of the License at
 *
 *       http://www.apache.org/licenses/LICENSE-2.0
 *
 *   Unless required by applicable law or agreed to in writing, software
 *   distributed under the License is distributed on an "AS IS" BASIS,
 *   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *   See the License for the specific language governing permissions and
 *   limitations under the License.
 */
#include <gpu_btree.h>
#include <stdlib.h>
#include <thrust/sequence.h>
#include <algorithm>
#include <cmd.hpp>
#include <cstdint>
#include <gpu_timer.hpp>
#include <numeric>
#include <random>
#include <rkg.hpp>
#include <string>
#include <unordered_set>
#include <validation.hpp>
#include <vector>

#include <device_bump_allocator.hpp>
#include <slab_alloc.hpp>

namespace cg = cooperative_groups;

template <typename Allocator, int B>
__global__ void kernel(Allocator allocator) {
  auto thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  auto block     = cg::this_thread_block();
  auto tile      = cg::tiled_partition<B>(block);
  auto tile_id   = thread_id / B;

  using allocator_type = device_allocator_context<Allocator>;
  allocator_type device_allocator{allocator, tile};
  const size_t num_to_allocate = 33554432;
  uint32_t ptrs[num_to_allocate];
  if (tile_id == 1) {
    for (size_t i = 0; i < num_to_allocate; i++) {
      ptrs[i] = device_allocator.allocate(allocator, 1, tile);
    }
    for (size_t i = 0; i < num_to_allocate; i++) {
      if (tile.thread_rank() == 0) { device_allocator.deallocate(allocator, ptrs[i], 0); }
    }
  }
}

template <int B = 16>
struct node {
  uint32_t pairs[B * 2];
};
int main(int argc, char** argv) {
  //{ SlabAllocLight<node_type, 13, 8, 32> a; }  // 8 GiBs
  //{ SlabAllocLight<node_type, 12, 8, 32> a; }  // 4 GiBs
  (void)argc;
  (void)argv;
  static constexpr int B = 16;
  using node_type        = node<B>;

  using slab_allocator_type = device_allocator::SlabAllocLight<node_type, 8, 1024 * 8, 16, 128>;

  slab_allocator_type allocator;

  allocator.get_allocated_count();

  kernel<slab_allocator_type, B><<<1, 64>>>(allocator);
  cuda_try(hipDeviceSynchronize());

  allocator.get_allocated_count();
}
